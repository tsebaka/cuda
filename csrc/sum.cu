
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__global__ void add(int a, int b, int *c) {
    *c = a + b;
}

int main() {
    int c;
    int *dev_c;
    hipMalloc((void**)&dev_c, sizeof(int));
    
    add<<<1, 1>>>(2, 7, dev_c);
    
    hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "2 + 7 = " << c;
    hipFree(dev_c);
    
    return 0;
}
